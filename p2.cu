#include "hip/hip_runtime.h"
#include "helper.hpp"
#include <hip/hip_runtime.h>
#include <chrono>


// cudaCalloc macro with error checking
#define cudaCalloc(A, B, C) \
  do { \
    hipError_t __cudaCalloc_err = hipMalloc(A, B*C); \
    if (__cudaCalloc_err == hipSuccess) hipMemset(*A, 0, B*C); \
  } while (0)

__global__ void p2_kernel(double *f4_i,
 double *c3,
 mwIndex *jStart,
 mwIndex *ii,
 double *fl,
 int *isNgbh,
 mwIndex *isUsed)
{
    // Calculate the starting and ending indices of the nonzero elements in the current row
    mwIndex i = blockIdx.x * blockDim.x + threadIdx.x;
    mwIndex start = jStart[i];
    mwIndex end = jStart[i + 1];
    mwIndex count = end - start;

    // Allocate shared memory for the pos array
    extern __shared__ int pos[];

    // Initialize cnt to 0
    mwIndex cnt = 0;

    // Loop through every nonzero element A(i,k)
    for (mwIndex id_i = start + threadIdx.x; id_i < end; id_i += blockDim.x){

        // get the column (k)
        mwIndex k = ii[id_i];

        isNgbh[k] = id_i+1;

        // loop through all nonzero elements A(k,j)
        for (mwIndex id_k = jStart[k]; id_k < jStart[k+1]; id_k++){

            // get the column (j)
            mwIndex j = ii[id_k];

            if (i == j) continue;

            // if this column is not visited yet for this row (i), then set it
            if (!isUsed[j]) {
                fl[j] = 0.0;  // initialize corresponding element
                isUsed[j] = 1;  // set column as visited
                pos[cnt++] = j;  // add column position to list of visited
            }

            // increase count of A(i,j)
            fl[j]++;

        }

    }

    // perform reduction on [cnt] non-empty columns (j)
    for (mwIndex l=0; l<cnt; l++) {

        // get next column number (j)
        int j = pos[l];

        if (isNgbh[j]) {
            c3[isNgbh[j]-1]  = fl[j];

            // atomic add to f4_i[0] to avoid race conditions
            atomicAdd(&f4_i[0], fl[j]);
        }

        // declare it non-used
        isUsed[j] = 0;
    }

    // divide by 2 only once in the first thread of each block
    if (threadIdx.x == 0) {
        f4_i[0] /= 2.0;
    }
}

void p2(double *f4, mwSize n, mwSize m, mwIndex *jStart, mwIndex *ii) {

    // start time measurement
    auto start = std::chrono::high_resolution_clock::now();

    // allocate device memory
    double *d_f4, *d_c3, *fl;
    mwIndex *d_jStart, *d_ii, *isUsed;
    int *isNgbh;
        
    hipMalloc(&d_jStart, (n + 1) * sizeof(int));
    hipMalloc(&d_ii, m * sizeof(int));

    cudaCalloc(&d_f4, n , sizeof(double) );
    cudaCalloc(&fl, n, sizeof(double) );
    cudaCalloc(&isNgbh, n, sizeof(int) );
    cudaCalloc(&d_c3, m, sizeof(double) );
    cudaCalloc(&isUsed, n, sizeof(mwIndex) );

    // copy input data to device memory
    hipMemcpy(d_ii, ii, (n + 1) * sizeof(mwIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_jStart, jStart, m * sizeof(mwIndex), hipMemcpyHostToDevice);

    // set up grid and block dimensions
    dim3 grid_dim(1,1);
    dim3 block_dim(n);
    if(n > 1024){
        grid_dim.x = (n + 1024 - 1) / 1024;
        block_dim.x = 1024;
    }

    std::cout << "Running cuda kernels\n";
    // launch the CUDA kernel
    p2_kernel<<<grid_dim, block_dim>>>(d_f4, d_c3, d_jStart, d_ii, fl, isNgbh, isUsed);

    // synchronisation
    hipDeviceSynchronize();

    // copy output data back to host memory
    hipMemcpy(f4, d_f4, sizeof(double), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_f4);
    hipFree(d_c3);
    hipFree(d_jStart);
    hipFree(d_ii);
    
    // stop time measurement
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Time taken by p2: " << duration << " milliseconds" << std::endl;
}